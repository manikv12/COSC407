#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void test(int *thr, int n);

int main()
{
    int n = 100;
    int *resThr;
    int *memThr = (int*)calloc(n*n,sizeof(int));
    memset(memThr, 0, sizeof(int)*(n*n));
    hipMalloc((void **) &resThr, sizeof(int) * (n*n));
    hipMemset(resThr, 0, sizeof(int)*(n*n));
    test<<<n,n,1>>>(resThr, n*n);
    hipDeviceSynchronize();
    hipMemcpy(memThr,resThr,sizeof(int)*(n*n),hipMemcpyDeviceToHost);
    hipFree(resThr);
    int i;
    for(i=0;i<(n*n);i++)
    {
        printf("%d, ",memThr[i]);
    }
}
__global__ void test(int *thr, int n)
{
    int idx = (blockDim.x*blockIdx.x)+threadIdx.x;
    
    if(idx <= n) 
        thr[idx] = idx;
}


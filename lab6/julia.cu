#include "hip/hip_runtime.h"
/*
Attribution to http://rosettacode.org/wiki/Bitmap/Write_a_PPM_file
for help with writing to a .ppm file, and to 
https://github.com/smithbower/julia for help with the algorithm for the julia set.
*/


#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#define WIDTH 2000
#define HEIGHT 2000
#define NEWTON_IT 100
#define ZOOM 2.2f
#define EPSILON 0.01f
double f(double z);
void cpu_julia(int *matrix);
void gpu_julia_setup();

__device__ __host__ void complex_add(float a, float b, float c, float d, float *realOut, float *imgOut)
{
    *realOut = a + c;
    *imgOut = b + d;
}

__device__ __host__ void complex_sub(float a, float b, float c, float d, float *realOut, float *imgOut)
{
    *realOut = a - c;
    *imgOut = b - d;
}

__device__ __host__ void complex_mul(float a, float b, float c, float d, float *realOut, float *imgOut)
{
    *realOut = (a * c) - (b * d);
    *imgOut = (b * c) + (a * d);
}

__device__ __host__ void complex_div(float a, float b, float c, float d, float *realOut, float *imgOut)
{
    *realOut = ((a * c) + (b * d)) / (pow(c, 2) + pow(d, 2));
    *imgOut = ((b * c) - (a * d))/ (pow(c, 2) + pow(d, 2));
}

__global__ void gpu_julia(int *matrix, int height, int width, int iterations, float epsilon, float zoom)
{
    int threadID = (blockIdx.x * blockDim.x) + threadIdx.x;
    float newRe, newIm, oldRe, oldIm;
    float z_3_r, z_3_i, z_2_r, z_2_i, inner_r, inner_i;
    if(threadID < (height * width))
    {
        int x = (threadID / height);
        int y = (threadID % height);
        newRe = (((float)x / (float)width) - 0.5f) * 2.0f * zoom;
        newIm = ((float)height / (float)width) * (((float)y / (float)height) - 0.5f) * 2.0f * zoom;
        int i;
        for(i = 0; i < iterations; i++)
        {
            oldRe = newRe;
            oldIm = newIm;
            //Clear everything.
            z_3_r = z_3_i = z_2_r = z_2_i = inner_r = inner_i = 0;
            complex_mul(oldRe, oldIm, oldRe, oldIm, &z_2_r, &z_2_i); // z^2
            complex_mul(z_2_r, z_2_i, oldRe, oldIm, &z_3_r, &z_3_i); // z^3
            z_3_r -= 1.0f; //z^3 - 1
            z_2_r *= 3.0f; // 3z^2
            z_2_i *= 3.0f;
            complex_div(z_3_r, z_3_i, z_2_r, z_2_i, &inner_r, &inner_i); // ((z^3 - 1) / 3z^2)
            complex_sub(oldRe, oldIm, inner_r, inner_i, &newRe, &newIm); //z - ((z^3 - 1) / 3z^2)
            //If we've mostly converged, break out early.
            if (abs(newRe - oldRe) < epsilon && abs(newIm - oldIm) < epsilon)
                break;
        }
            if (abs(1.0f - newRe) < epsilon && abs(0 - newIm) < epsilon)
                matrix[threadID] = 1;
            else
            if (newRe - 0.5f < epsilon && 0.86603f - newIm < epsilon)
                matrix[threadID] = 2;
            else
            if (newRe - 0.5f < epsilon && newIm - 0.86603f < epsilon)
                matrix[threadID] = 3;
            else
                matrix[threadID] = 0;
    }
}

__host__ void gpu_julia_setup()
{
    int *gpu_device_matrix;
    int *gpu_matrix = (int*)malloc(sizeof(int) * WIDTH * HEIGHT);
    hipMalloc((void **)&gpu_device_matrix, sizeof(int) * HEIGHT * WIDTH);
    gpu_julia<<<(WIDTH*HEIGHT/128),128>>>(gpu_device_matrix, HEIGHT, WIDTH, NEWTON_IT, EPSILON, ZOOM);
    hipDeviceSynchronize();
    hipMemcpy(gpu_matrix, gpu_device_matrix, (sizeof(int) * HEIGHT * WIDTH), hipMemcpyDeviceToHost); 
    //cpu_julia(gpu_matrix);

    int i, j;
    FILE *fp = fopen("julia.ppm", "wb"); /* b - binary mode */
    (void) fprintf(fp, "P6\n%d %d\n255\n", WIDTH, HEIGHT);
    for (j = 0; j < HEIGHT; ++j)
    {
        for (i = 0; i < WIDTH; ++i)
        {
            unsigned char color[3];
            int val = gpu_matrix[j*HEIGHT +i];
            switch(val)
            {
                case 1:
                    color[0] = 180;
                    color[1] = 82;
                    color[2] = 205;
                break;
                case 2:
                    color[0] = 255;
                    color[1] = 48;
                    color[2] = 48;
                    break;
                case 3:
                    color[0] = 0; 
                    color[1] = 205;
                    color[2] = 0;
                break;
                default:
                    color[0] = 0;
                    color[1] = 0;
                    color[2] = 205;              
                    break;
            }
            (void) fwrite(color, 1, 3, fp);
        }
    }
    (void) fclose(fp);   
}

double f(double z)
{
    double g = pow(z, 3.0f) - 1.0f;
    double g_p = 3.0f * pow(z,2.0f);
    double f_z = z - ((double)g/(double)g_p);
    return f_z;  
}

 
void cpu_julia(int *matrix)
{
    int x,y,i;
    float newRe, newIm, oldRe, oldIm;
    float z_3_r, z_3_i, z_2_r, z_2_i, inner_r, inner_i;
    float ratio = (float)HEIGHT / (float)WIDTH;
    
    for(x = 0; x < WIDTH; x++)
    {
        for(y = 0; y < HEIGHT; y++)
        {
            newRe = (((float)x / (float)WIDTH) - 0.5f) * 2.0f * ZOOM;
            newIm = ratio * (((float)y / (float)HEIGHT) - 0.5f) * 2.0f * ZOOM;
            for(i = 0; i < NEWTON_IT; i++)
            {
                oldRe = newRe;
                oldIm = newIm;

                //Clear everything.
                z_3_r = z_3_i = z_2_r = z_2_i = inner_r = inner_i = 0;

                complex_mul(oldRe, oldIm, oldRe, oldIm, &z_2_r, &z_2_i); // z^2
                complex_mul(z_2_r, z_2_i, oldRe, oldIm, &z_3_r, &z_3_i); // z^3
                z_3_r -= 1.0f; //z^3 - 1

                z_2_r *= 3.0f; // 3z^2
                z_2_i *= 3.0f;

                complex_div(z_3_r, z_3_i, z_2_r, z_2_i, &inner_r, &inner_i); // ((z^3 - 1) / 3z^2)

                complex_sub(oldRe, oldIm, inner_r, inner_i, &newRe, &newIm); //z - ((z^3 - 1) / 3z^2)

                //If we've mostly converged, break out early.
                if (abs(newRe - oldRe) < EPSILON && abs(newIm - oldIm) < EPSILON)
                    break;
            }
            if (abs(1.0f - newRe) < EPSILON && abs(0 - newIm) < EPSILON)
                matrix[x * HEIGHT + y] = 1;
            else
            if (newRe - 0.5f < EPSILON && 0.86603f - newIm < EPSILON)
                matrix[x * HEIGHT + y] = 2;
            else
            if (newRe - 0.5f < EPSILON && newIm - 0.86603f < EPSILON)
                matrix[x * HEIGHT + y] = 3;
            else
                matrix[x * HEIGHT + y] = 0;
        }
    }
}

int main()
{
    int *matrix = (int*)malloc(sizeof(int) * WIDTH * HEIGHT);
    cpu_julia(matrix);
    gpu_julia_setup();
    return 0;
}  

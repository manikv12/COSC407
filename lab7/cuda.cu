#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
float h(float a, float b, int n);
__device__ float f_dev(float x, float y);
float f(float x, float y);

typedef struct{
    float val;
    float x;
    float y;
} point;

__global__ void f2(float hx, float hy, float xa, float ya, point *memBlock, int points);
point *findMin(point *Points, int size);

int main()
{
    int n = 100;
    float eps = 0.000001;
    float hx, hy, xa = -1.0, xb = 1.0, ya = -1.0, yb = 1.0;
    while((h(xa, xb, n) >= eps) || (h(ya, yb, n) >= eps))
    {
        point* Points = (point*)malloc(sizeof(point)*(n*n));
        point* devPoints;
        
        hipMalloc(&devPoints, sizeof(point)*(n*n));
        
        hx = h(xa, xb, n);
        hy = h(ya, yb, n);
        
        dim3 threads(1, 1);
        dim3 block(n, n);
        f2<<<block, threads>>>(hx, hy, xa, ya, devPoints, (n*n)); 
        
        hipDeviceSynchronize();
        hipMemcpy(Points, devPoints, sizeof(point)*(n*n), hipMemcpyDeviceToHost);

        point* Min = findMin(Points, n*n);
        
        xa = ((Min->x)-hx);
        xb = ((Min->x)+hx);
        ya = ((Min->y)-hy);
        yb = ((Min->y)+hy);

        hipFree(devPoints);
        free(Points);
        free(Min);
    }
    float minx = (xa + xb)/2.0f;
    float miny = (ya + yb)/2.0f;
    //printf("xa: %f\nxb: %f\nya: %f\nyb: %f\n", xa, xb, ya, yb);
    printf("CUDA global min: x = %f, y = %f, f = %f\n", minx, miny,f(minx,miny));
    return 0;
}

__global__ void f2(float hx, float hy, float xa, float ya, point* memBlock, int points)
{
    int idx = (blockDim.x*blockIdx.x)+threadIdx.x;
    if(idx < (points)) 
    {
        memBlock[idx].val = f_dev(xa+(hx*blockIdx.x), ya+(hy*threadIdx.x));
        memBlock[idx].x = (xa + (hx*blockIdx.x));
        memBlock[idx].y = (ya + (hy*threadIdx.x));
    }
}

point* findMin(point* Points, int size)
{
    point *Min = (point *)malloc(sizeof(point));
    Min->val = Points[0].val;
    Min->x = Points[0].x;
    Min->y = Points[0].y;
    int i;
    for(i = 1;i < size;i++)
    {
        if((Points[i].val) < (Min->val))
        {
            Min->val = Points[i].val;
            Min->x = Points[i].x;
            Min->y = Points[i].y;
        }
    }
    return Min;
}

float f(float x, float y)
{
    return(x*x*(4-2.1*x*x+(x*x*x*x)/3)+x*y+y*y*(-4+4*y*y));
}

float h(float a, float b, int n)
{
    return ((b-a)/(float)n);
}
__device__ float f_dev(float x, float y)
{
    return(x*x*(4-2.1*x*x+(x*x*x*x)/3)+x*y+y*y*(-4+4*y*y));
}


#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#define RED  "\x1B[31m"
#define GRN  "\x1B[32m"
#define CYN  "\x1B[36m"
#define WHT  "\x1B[37m"
__global__ void edt(int N, int count, int *dev_arr, int *matrix);
int main()
{
    int matrix1[5][5] = {{1,1,1,1,1},{1,1,1,1,1},{1,1,0,1,1},{1,1,1,1,1},{1,1,1,1,1}};
    //int matrix2[5][5] = {{1,1,1,1,1},{1,0,1,1,1},{1,1,0,1,1},{1,1,1,1,1},{1,1,1,1,1}};
    //int matrix3[5][5] = {{1,1,1,1,1},{1,0,1,0,1},{1,1,1,1,1},{1,1,0,1,1},{1,1,1,1,1}};
    int squared_matrix1[5][5];
    //int squared_matrix2[5][5];
    //int squared_matrix3[5][5];
    int a,b, N = 5, count = 0;
    for(a=0;a<N;a++)
    {
        for(b=0;b<N;b++)
        {
            squared_matrix1[a][b] = 1000;
            //squared_matrix2[a][b] = 1000;
            //squared_matrix3[a][b] = 1000;
        }
    }
    
    printf("\nCUDA Implementation\n");

    /* Count number of 0's in matrix */
    for(a=0;a<N;a++)
    {
        for(b=0;b<N;b++)
        {
            if(0==matrix1[a][b])
                count++;
        }
    }
    
    int *host_array = (int*)malloc(count*sizeof(int)*(N*N)); 
    int *dev_array, *matrix;
    hipMalloc(&dev_array, count*sizeof(int)*(N*N));
    hipMalloc(&matrix, sizeof(matrix1));
    hipMemcpy((void **)matrix, matrix1, sizeof(matrix1), hipMemcpyHostToDevice);
    edt<<<N,N>>>(N, count, dev_array, matrix);
    hipMemcpy(host_array, dev_array, count*sizeof(int)*(N*N), hipMemcpyDeviceToHost);
   

    /* TODO
        This will return an array containing 'count' matrices. It will then
        iterate through all of them at each index and set the least valued
        integer as the value at each index in squared_matrix
    */

    printf("\n"); 
    for(a=0;a<N;a++)
    {
        for(b=0;b<N;b++)
        {
            printf("%d ", host_array[a+b]);
        }
        printf("\n");
    }
    printf("\n");

    hipFree(dev_array);
    hipFree(matrix);
    free(host_array);
    return 0;
}

__global__ void edt(int N, int count, int *dev_arr, int *matrix)
{
    int idx = blockIdx.x+threadIdx.x;
    if(idx < (N*N))
    {
        int a,b;
        if(matrix[idx] == 0)
        {
            dev_arr[idx] = 10;
            for(a=0;a<N;a++)
            {
                for(b=0;b<N;b++)
                {
                    int tmp = abs(pow((float)(a-idx),2.0f)) + abs(pow((float)(b-idx),2.0f));
                    if((matrix[a+b] > tmp))
                    {
                        matrix[a+b] = 9;
                    }
                }
            }

        }
        else
            dev_arr[idx] = 9;
    }
}











